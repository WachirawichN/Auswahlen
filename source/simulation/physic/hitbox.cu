#include "hip/hip_runtime.h"
#include "hitbox.cuh"

std::vector<float> sphereHitbox(float position, float radius)
{
    return std::vector<float>({0.0f});
}
std::vector<float> hitbox::cubeHitbox(float position, float scale)
{
    // Return format: {-Axis, Origin, +Axis}
    std::vector<float> borders;
    borders.push_back(position - scale / 2.0f);
    borders.push_back(position);
    borders.push_back(position + scale / 2.0f);

    return borders;
}
std::vector<int> hitbox::hitboxSide(float objPos, float objVel, float tarPos, float tarVel, bool tarInObj, bool objInTar, float timeMultiplier)
{
    // Function will return -1, 1 so its need to be +1 before use
    int objToTarSide = mathExt::aToBside(objPos, tarPos);
    bool isObjFaster = abs(objVel) > abs(tarVel);
    int objVelDirection = mathExt::direction(objVel * timeMultiplier);
    int tarVelDirection = mathExt::direction(tarVel * timeMultiplier);

    // Hollow body
    if (objInTar)
    {
        // Opposite direction
        if (objVelDirection != tarVelDirection)
        {
            if (objVelDirection == mathExt::MIDDLE)
            {
                //std::cout << "Obj velocity is middle" << std::endl;
                return std::vector<int>({tarVelDirection, tarVelDirection});
            }
            //std::cout << "Obj velocity is not middle" << std::endl;
            return std::vector<int>({objVelDirection, objVelDirection});
        }

        // Same direction
        if (objVelDirection != mathExt::MIDDLE)
        {
            //std::cout << "None is middle: " << convertNegative(isObjFaster) << ", " << tarVelDirection << std::endl;
            return std::vector<int>({mathExt::falseToMinus(isObjFaster) * objVelDirection, mathExt::falseToMinus(isObjFaster) * objVelDirection});
        }
    }
    else if (tarInObj)
    {
        // Opposite direction
        if (objVelDirection != tarVelDirection)
        {
            if (tarVelDirection == mathExt::MIDDLE)
            {
                //std::cout << "Tar velocity is middle" << std::endl;
                return std::vector<int>({objVelDirection, objVelDirection});
            }
            //std::cout << "Tar velocity is not middle" << std::endl;
            return std::vector<int>({tarVelDirection, tarVelDirection});
        }

        // Same direction
        if (tarVelDirection != mathExt::MIDDLE)
        {
            //std::cout << "None is middle: " << convertNegative(isObjFaster) << ", " << tarVelDirection << std::endl;
            return std::vector<int>({mathExt::falseToMinus(!isObjFaster) * tarVelDirection, mathExt::falseToMinus(!isObjFaster) * tarVelDirection});
        }
    }

    // Solid body
    //std::cout << "Solid body" << std::endl;
    return std::vector<int>({objToTarSide, objToTarSide * -1});
}