#include "hip/hip_runtime.h"
#include "collision.cuh"

float timeToMove(float aPosition, float aVelocity, float bPosition, float bVelocity)
{
    float totalVelocity = aVelocity - bVelocity;
    float dst = bPosition - aPosition;
    return (totalVelocity != 0.0f) ? (dst / totalVelocity) : 999999.99f;
}
std::vector<float> unbalanceTimeToMove(float aPosition, float aVelocity, float aRemainTime, float bPosition, float bVelocity, float bRemainTime)
{
    unsigned int lowestTime = (aRemainTime < bRemainTime) ? 0 : 1;
    return std::vector<float>({0.0f});
}

bool isInside(std::vector<float> aBorders, std::vector<float> bBorders)
{
    bool positiveSide = (bBorders[2] < aBorders[2]) && (bBorders[0] < aBorders[2]);
    bool negativeSide = (bBorders[2] > aBorders[0]) && (bBorders[0] > aBorders[0]);

    return (positiveSide && negativeSide);
}
bool isCross(std::vector<float> aBorders, std::vector<float> bBorders)
{
    bool positiveSide = ((bBorders[0] < aBorders[2]) && !(bBorders[2] < aBorders[2])) || (!(bBorders[0] < aBorders[2]) && (bBorders[2] < aBorders[2]));
    bool negativeSide = ((bBorders[2] < aBorders[0]) && !(bBorders[0] < aBorders[0])) || (!(bBorders[2] < aBorders[0]) && (bBorders[0] < aBorders[0]));

    return (positiveSide || negativeSide);
}

std::vector<std::vector<unsigned int>> collision::collisionPairing(std::vector<std::shared_ptr<object::objectBaseClass>> objects)
{
    // The algorithm should sort the pairing base on the distance of each pair

    // Object at index 0 is the current object then after that is all the target object
    std::vector<std::vector<unsigned int>> pairs;

    for (unsigned int i = 0; i < objects.size(); i++)
    {
        std::shared_ptr<object::objectBaseClass> currentObject = objects.at(i);
        if (!currentObject->canCollide()) continue;
        std::vector<unsigned int> currentPair = {i};

        for (unsigned int j = i + 1; j < objects.size(); j++)
        {
            std::shared_ptr<object::objectBaseClass> targetObject = objects.at(j);
            if (!targetObject->canCollide()) continue;
            currentPair.push_back(j);
        }

        if (currentPair.size() >= 1)
        {
            pairs.push_back(currentPair);
        }
    }

    return pairs;
}
std::vector<collision::collisionType> collision::CCD(std::shared_ptr<object::objectBaseClass> obj, std::shared_ptr<object::objectBaseClass> tar, float deltaTime)
{
    std::vector<collision::collisionType> collisionResults;
    int timeMultiplier = (deltaTime > 0.0f) ? 1 : -1;

    glm::vec3 objPos = obj->getPosition();
    glm::vec3 objScale = obj->getScale();
    glm::vec3 objVel = obj->getVelocity();
    float objectDeltaTime = obj->getCollisionTime();

    glm::vec3 tarPos = tar->getPosition();
    glm::vec3 tarScale = tar->getScale();
    glm::vec3 tarVel = tar->getVelocity();
    float targetDeltaTime = obj->getCollisionTime();

    std::cout << "   -  Selected border:" << std::endl;
    for (int axis = 0; axis < 3; axis++)
    {
        // Calculate the border of both object
        std::vector<float> objBorders = hitbox::cubeHitbox(objPos[axis], objScale[axis]);
        std::vector<float> tarBorders = hitbox::cubeHitbox(tarPos[axis], tarScale[axis]);

        // Check if the object is inside each other or not
        bool tarInObj = isInside(objBorders, tarBorders);
        bool objInTar = isInside(tarBorders, objBorders);


        // Choose object border
        std::vector<int> selectedBorder = hitbox::hitboxSide(objPos[axis], objVel[axis], tarPos[axis], tarVel[axis], tarInObj, objInTar, timeMultiplier);
        int objBorderIndex = selectedBorder[0] + 1;
        int tarBorderIndex = selectedBorder[1] + 1;

        float objBorder = objBorders[objBorderIndex];
        float tarBorder = tarBorders[tarBorderIndex];
        std::cout << "      -  Axis: " << axis << ", object's: " << objBorderIndex << ", target's: " << tarBorderIndex << std::endl;

        // Check for NEWLY collision type
        float travelTime = timeToMove(objBorder, objVel[axis], tarBorder, tarVel[axis]);
        if (abs(deltaTime) > 0.0f && mathExt::roundToDec(abs(travelTime), 6) > 0.0f && abs(deltaTime) > abs(travelTime))
        {
            if (mathExt::direction(deltaTime) == mathExt::direction(travelTime))
            {
                //std::cout << "Travel time: " << mathExt::roundToDec(travelTime, 6) << ", " << (abs(travelTime) > 0.0f) << std::endl;
                collisionResults.push_back(collision::collisionType::NEWLY);
                std::cout << "         -  Distance: " << tarBorder - objBorder << ", Velocity: " << objVel[axis] - tarVel[axis] << std::endl;
                continue;
            }
        }

        // Check for CROSS and INSIDE colision type
        if (objInTar || tarInObj)
        {
            collisionResults.push_back(collision::collisionType::INSIDE);
            continue;
        }
        else if (isCross(objBorders, tarBorders))
        {
            collisionResults.push_back(collision::collisionType::CROSS);
            continue;
        }

        // No collision has been occure
        collisionResults.push_back(collision::collisionType::NO);
    }

    return collisionResults;
}
float collision::collisionResolver(std::shared_ptr<object::objectBaseClass> obj, std::shared_ptr<object::objectBaseClass> tar, float deltaTime, std::vector<unsigned int> newlyAxis)
{
    // Working instruction:
    // -  Step 1:
    //    -  Find the time that two object need to use to travel to collision site in all axis that is NEWLY collide axis.
    // -  Step 2:
    //    -  Put those travel times through pythagoras to get final time to move before collision will be occured.
    // -  Step 3:
    //    -  Move those object to collision site.
    // -  Step 4:
    //    -  Update object and target velocity that is NEWLY collide axis.

    int timeMultiplier = (deltaTime > 0.0f) ? 1 : -1;

    glm::vec3 objPos = obj->getPosition();
    glm::vec3 objScale = obj->getScale();
    glm::vec3 objVel = obj->getVelocity();

    glm::vec3 tarPos = tar->getPosition();
    glm::vec3 tarScale = tar->getScale();
    glm::vec3 tarVel = tar->getVelocity();

    // Step 1.
    std::vector<float> travelTimes;
    for (unsigned int axis : newlyAxis)
    {
        // Calculate the border of both object
        std::vector<float> objBorders = hitbox::cubeHitbox(objPos[axis], objScale[axis]);
        std::vector<float> tarBorders = hitbox::cubeHitbox(tarPos[axis], tarScale[axis]);

        // Check if the object is inside each other or not
        bool tarInObj = isInside(objBorders, tarBorders);
        bool objInTar = isInside(tarBorders, objBorders);

        // Choose object border
        std::vector<int> selectedBorder = hitbox::hitboxSide(objPos[axis], objVel[axis], tarPos[axis], tarVel[axis], tarInObj, objInTar, timeMultiplier);
        int objMultiplier = selectedBorder[0] + 1;
        int tarMultiplier = selectedBorder[1] + 1;

        float objBorder = objBorders[objMultiplier];
        float tarBorder = tarBorders[tarMultiplier];

        float travelTime = timeToMove(objBorder, objVel[axis], tarBorder, tarVel[axis]);
        travelTimes.push_back(travelTime);
    }

    // Step 2.
    float lowestTime = *std::min_element(travelTimes.begin(), travelTimes.end());

    // Step 3.
    obj->move(fundamental::calculateDst(objVel, lowestTime));
    tar->move(fundamental::calculateDst(tarVel, lowestTime));

    // Step 4.
    glm::vec3 objDeltaVel(0.0f);
    glm::vec3 tarDeltaVel(0.0f);
    std::cout << "      -  Pythagoras time: " << lowestTime << std::endl;
    std::cout << "      -  Resolving collision" << std::endl;
    for (unsigned int axis : newlyAxis)
    {
        float objNewVel = momentum::elasticCollision1D(obj, tar, axis);
        float tarNewVel = momentum::elasticCollision1D(tar, obj, axis);

        tarDeltaVel[axis] = tarNewVel - tarVel[axis];
        objDeltaVel[axis] = objNewVel - objVel[axis];

        std::cout << "         -  Axis: " << axis << std::endl;
        std::cout << "            -  Axis travel time: " << travelTimes[axis] << std::endl;
        std::cout << "            -  Object original axis vel: " << objVel[axis] << ", new axis vel: " << objNewVel << std::endl;
        std::cout << "            -  Target original axis vel: " << tarVel[axis] << ", new axis vel: " << tarNewVel << std::endl;
        std::cout << "            -  Object original axis pos: " << objPos[axis] << ", new axis pos: " << obj->getPosition()[axis] << std::endl;
        std::cout << "            -  Target original axis pos: " << tarPos[axis] << ", new axis pos: " << tar->getPosition()[axis] << std::endl;
    }
    obj->changeVelocity(objDeltaVel);
    tar->changeVelocity(tarDeltaVel);

    return lowestTime;
}
