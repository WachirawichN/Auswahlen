#include "hip/hip_runtime.h"
#include "objectBaseClass.cuh"

#include <iostream>

glm::vec3 normalizeRotation(glm::vec3 objectRotation)
{
    glm::vec3 newRotation(objectRotation.x, objectRotation.y, objectRotation.z);
    for (int i = 0; i < 3; i++)
    {
        if (objectRotation[i] >= 360 || objectRotation[i] < 0)
        {
            int multiplier = floor(objectRotation[i] / 360.0f);
            newRotation[i] -= multiplier * 360.0f;
        }
    }
    return newRotation;
}

object::objectBaseClass::objectBaseClass(bool collision, bool anchored, float mass, glm::vec3 velocity, glm::vec3 position, glm::vec3 rotation, glm::vec3 scale, glm::vec4 color)
    : collision(collision), anchored(anchored), mass(mass), velocity(velocity), collisionTime(float(0.0f)), position(position), rotation(normalizeRotation(rotation)), scale(scale), color(color)
{
}

__host__ __device__ bool object::objectBaseClass::canCollide()
{
    return collision;
}
__host__ __device__ bool object::objectBaseClass::isAnchored()
{
    return anchored;
}

void object::objectBaseClass::setCollision(bool newValue)
{
    collision = newValue;
}
void object::objectBaseClass::setAnchored(bool newValue)
{
    anchored = newValue;
}

__host__ __device__ void object::objectBaseClass::changeVelocity(glm::vec3 deltaVelocity)
{
    velocity += deltaVelocity;
}
__host__ __device__ void object::objectBaseClass::changeCollisionTime(float deltaTime)
{
    collisionTime += deltaTime;
}

__host__ __device__ void object::objectBaseClass::move(glm::vec3 distance)
{
    position += distance;
}
__host__ __device__ void object::objectBaseClass::rotate(glm::vec3 degree)
{
    rotation += degree;
    rotation = normalizeRotation(rotation); // Make the rotation be within 360 degree
}
__host__ __device__ void object::objectBaseClass::rescale(glm::vec3 deltaScale)
{
    scale = deltaScale;
}

void object::objectBaseClass::setColor(glm::vec4 newColor)
{
    color = newColor;
}

__host__ __device__ float object::objectBaseClass::getMass() const
{
    return mass;
}
__host__ __device__ glm::vec3 object::objectBaseClass::getVelocity() const
{
    return velocity;
}
__host__ __device__ float object::objectBaseClass::getCollisionTime() const
{
    return collisionTime;
}

__host__ __device__ glm::vec3 object::objectBaseClass::getPosition() const
{
    return position;
}
__host__ __device__ glm::vec3 object::objectBaseClass::getRotation() const
{
    return rotation;
}
__host__ __device__ glm::vec3 object::objectBaseClass::getScale() const
{
    return scale;
}

glm::vec4 object::objectBaseClass::getColor() const
{
    return color;
}